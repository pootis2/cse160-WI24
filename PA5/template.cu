#include "hip/hip_runtime.h"
#include <gputk.h>

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE
__global__ void convolution(float *image, const float * __restrict__ mask, float *out, int channels, int width, int height){

    __shared__ float subInput[w][w][3];

    int tx = threadIdx.x; int bx = blockIdx.x;
    int ty = threadIdx.y; int by = blockIdx.y; 
    int tz = threadIdx.z;

    // output row col
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    // input start row col
    int row1 = row - Mask_radius;
    int col1 = col - Mask_radius;

    if(row1 >= 0 && row1 < height && col1 >= 0 && col1 < width){
        subInput[ty][tx][tz] = image[(row1*width + col1)*3 + tz];
    }
    else{
        subInput[ty][tx][tz] = 0;
    }
    __syncthreads();
    float ans = 0;
    if(ty < TILE_WIDTH && tx < TILE_WIDTH && row < height && col < width){
        for(int i = 0; i < Mask_width; i++){
            for(int j = 0; j < Mask_width; j++){
                ans += mask[i*Mask_width + j] * subInput[ty + i][tx + j][tz];
            }
        }
        out[(row * width + col)*3 + tz] = clamp(ans);
    }

}

int main(int argc, char *argv[]) {
  gpuTKArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  gpuTKImage_t inputImage;
  gpuTKImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = gpuTKArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = gpuTKArg_getInputFile(arg, 0);
  inputMaskFile  = gpuTKArg_getInputFile(arg, 1);

  inputImage   = gpuTKImport(inputImageFile);
  hostMaskData = (float *)gpuTKImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
  assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

  imageWidth    = gpuTKImage_getWidth(inputImage);
  imageHeight   = gpuTKImage_getHeight(inputImage);
  imageChannels = gpuTKImage_getChannels(inputImage);

  outputImage = gpuTKImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = gpuTKImage_getData(inputImage);
  hostOutputImageData = gpuTKImage_getData(outputImage);

  gpuTKTime_start(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKTime_start(GPU, "Doing GPU memory allocation");
  //@@ INSERT CODE HERE
  hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **) &deviceMaskData, maskColumns * maskRows * sizeof(float));

  gpuTKTime_stop(GPU, "Doing GPU memory allocation");

  gpuTKTime_start(Copy, "Copying data to the GPU");
  //@@ INSERT CODE HERE
  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceOutputImageData, hostOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceMaskData, hostMaskData, maskColumns * maskRows * sizeof(float), hipMemcpyHostToDevice);

  gpuTKTime_stop(Copy, "Copying data to the GPU");

  gpuTKTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 dimBlock(w, w, imageChannels); 
  dim3 dimGrid(ceil((float)imageWidth/TILE_WIDTH), ceil((float)imageHeight/TILE_WIDTH)); 
  convolution<<<dimGrid, dimBlock>>>(hostInputImageData, hostMaskData,
                                     hostOutputImageData, imageChannels,
                                     imageWidth, imageHeight);

  gpuTKTime_stop(Compute, "Doing the computation on the GPU");

  gpuTKTime_start(Copy, "Copying data from the GPU");
  //@@ INSERT CODE HERE
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost);
  gpuTKTime_stop(Copy, "Copying data from the GPU");

  gpuTKTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKSolution(arg, outputImage);

  //@@ Insert code here
  hipFree(deviceInputImageData);
  hipFree(deviceMaskData);
  hipFree(deviceOutputImageData);

  free(hostMaskData);
  gpuTKImage_delete(outputImage);
  gpuTKImage_delete(inputImage);

  return 0;
}
